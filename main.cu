#define CATCH_CONFIG_RUNNER
#include <catch2/catch.hpp>

#include <hip/hip_runtime.h>

#include <iostream>


int main(int argc, char *argv[])
{
  Catch::Session session;

  int device_id {};

  // Build a new parser on top of Catch's
  using namespace Catch::clara;
  auto cli = session.cli()
             | Opt(device_id, "device")["-d"]["--device"]("device id to use");
  session.cli(cli);

  int returnCode = session.applyCommandLine(argc, argv);
  if(returnCode != 0)
  {
    return returnCode;
  }

  return session.run(argc, argv);
}
